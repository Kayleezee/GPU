/*
 *
 * nullKernelAsync.cu
 *
 * Microbenchmark for throughput of asynchronous kernel launch.
 *
 * Build with: nvcc -I ../chLib <options> nullKernelAsync.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include "chTimer.h"


int
main( int argc, char *argv[] )
{
int mem=atoi(argv[1]);
int vergleich=atoi(argv[2]); // pinned (vergleich==1)  memmory or not (else)
int vergleich2=atoi(argv[3]); // H2D (vergleich2==1) or D2H (else) 
bool *dmem;
bool *hmem;

hipMalloc((void**)&dmem,mem*1000); //Allocate GPU memory

if (vergleich==1) hmem= (bool*) malloc(mem*1000); //Allocate CPU memory
else 	          hipHostMalloc ((void**) &hmem,mem*1000) ;		

	const int cIterations = 10000;
	
    fflush( stdout );

    chTimerTimestamp start, stop;

    chTimerGetTime( &start );
    for ( int i = 0; i < cIterations; i++ ) {
	if (vergleich2==1) hipMemcpy(hmem,dmem,mem*1000,hipMemcpyDeviceToHost);//transfer data
        else hipMemcpy(dmem,hmem,mem*1000,hipMemcpyHostToDevice);
    }

    chTimerGetTime( &stop );

    {
        double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
        double usPerLaunch = microseconds / (float) cIterations;

        printf( "%.2f\n", usPerLaunch );
    }

hipFree(dmem);
if (vergleich==1) free( hmem);
else hipFree(hmem);

    return 0;
}
