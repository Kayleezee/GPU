#include "hip/hip_runtime.h"
/**************************************************************************************************
 *
 *       Computer Engineering Group, Heidelberg University - GPU Computing Exercise 06
 *
 *                 Gruppe : gpucomp02
 *
 *                   File : main.cu
 *
 *                Purpose : Reduction
 *
 **************************************************************************************************/

#include <cmath>
#include <iostream>
#include <cstdlib>
#include <chCommandLine.h>
#include <chTimer.hpp>

const static int DEFAULT_MATRIX_SIZE = 1024;
const static int DEFAULT_BLOCK_DIM   =  128;

//
// Function Prototypes
//
void printHelp(char *);

//
// Reduction_Kernel
//
__global__ void
reduction_Kernel(int numElements, float* dataIn, float* dataOut)
{
    extern __shared__ float sPartArray[];

    const int tid = threadIdx.x;
	unsigned int elementId = blockIdx.x * (blockSize*2) + threadIdx.x;
	unsigned int gridSize = blockSize * 2 * gridDim.x;

    sPartArray[tid] = 0;

	if (elementId < numElements)
	{
		/*TODO Kernel Code*/
		while(elementId < numElements) {
            sPartArray[tid] += dataIn[elementId] + dataIn[elementId + blockSize];
            elementId += gridSize;
		}
		__syncthreads();
	}

	if (tid == 0) {
        dataOut[blockIdx.x] = sPartArray[0];
	}
}

//
// Main
//
int
main(int argc, char * argv[])
{
	bool showHelp = chCommandLineGetBool("h", argc, argv);
	if (!showHelp)
	{
		showHelp = chCommandLineGetBool("help", argc, argv);
	}

	if (showHelp)
	{
		printHelp(argv[0]);
		exit(0);
	}

	std::cout << "***" << std::endl
			  << "*** Starting ..." << std::endl
			  << "***" << std::endl;

	ChTimer memCpyH2DTimer, memCpyD2HTimer;
	ChTimer kernelTimer;

	//
	// Allocate Memory
	//
	int numElements = 0;
	chCommandLineGet<int>(&numElements, "s", argc, argv);
	chCommandLineGet<int>(&numElements, "size", argc, argv);
	numElements = numElements != 0 ?
			numElements : DEFAULT_MATRIX_SIZE;
	//
	// Host Memory
	//
	bool pinnedMemory = chCommandLineGetBool("p", argc, argv);
	if (!pinnedMemory)
	{
		pinnedMemory = chCommandLineGetBool("pinned-memory",argc,argv);
	}

	float* h_dataIn = NULL;
	float* h_dataOut = NULL;
	if (!pinnedMemory)
	{
		// Pageable
		h_dataIn = static_cast<float*>
				(malloc(static_cast<size_t>(numElements * sizeof(*h_dataIn))));
		h_dataOut = static_cast<float*>
				(malloc(static_cast<size_t>(sizeof(*h_dataOut))));
	}
	else
	{
		// Pinned
		hipHostMalloc(&h_dataIn,
				static_cast<size_t>(numElements * sizeof(*h_dataIn)));
		hipHostMalloc(&h_dataOut,
				static_cast<size_t>(sizeof(*h_dataOut)));
	}
	// Init h_dataOut
	*h_dataOut = 0;

	// Device Memory
	float* d_dataIn = NULL;
	float* d_dataOut = NULL;
	hipMalloc(&d_dataIn,
			static_cast<size_t>(numElements * sizeof(*d_dataIn)));
	hipMalloc(&d_dataOut,
			static_cast<size_t>(sizeof(*d_dataOut)));

	if (h_dataIn == NULL || h_dataOut == NULL ||
		d_dataIn == NULL || d_dataOut == NULL)
	{
		std::cout << "\033[31m***" << std::endl
		          << "*** Error - Memory allocation failed" << std::endl
		          << "***\033[0m" << std::endl;

		exit(-1);
	}

	//
	// Copy Data to the Device
	//
	memCpyH2DTimer.start();

	hipMemcpy(d_dataIn, h_dataIn,
			static_cast<size_t>(numElements * sizeof(*d_dataIn)),
			hipMemcpyHostToDevice);
	hipMemcpy(d_dataOut, h_dataOut,
			static_cast<size_t>(sizeof(*d_dataOut)),
			hipMemcpyHostToDevice);

	memCpyH2DTimer.stop();

	//
	// Get Kernel Launch Parameters
	//
	int blockSize = 0,
		gridSize = 0;

	// Block Dimension / Threads per Block
	chCommandLineGet<int>(&blockSize,"t", argc, argv);
	chCommandLineGet<int>(&blockSize,"threads-per-block", argc, argv);
	blockSize = blockSize != 0 ?
			blockSize : DEFAULT_BLOCK_DIM;

	if (blockSize > 1024)
	{
		std::cout << "\033[31m***" << std::endl
		          << "*** Error - The number of threads per block is too big" << std::endl
		          << "***\033[0m" << std::endl;

		exit(-1);
	}

	gridSize = ceil(static_cast<float>(numElements) / static_cast<float>(blockSize));

	dim3 grid_dim = dim3(gridSize);
	dim3 block_dim = dim3(blockSize);

	kernelTimer.start();

	reduction_Kernel<<<grid_dim, block_dim>>>(numElements, d_dataIn, d_dataOut);

	// Synchronize
	hipDeviceSynchronize();

	// Check for Errors
	hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess)
	{
		std::cout << "\033[31m***" << std::endl
				  << "***ERROR*** " << hipError_t << " - " << hipGetErrorString(hipError_t)
				  	<< std::endl
				  << "***\033[0m" << std::endl;

		return -1;
	}

	kernelTimer.stop();

	//
	// Copy Back Data
	//
	memCpyD2HTimer.start();

	hipMemcpy(h_dataOut, d_dataOut,
			static_cast<size_t>(sizeof(*d_dataOut)),
			hipMemcpyDeviceToHost);

	memCpyD2HTimer.stop();

	// Free Memory
	if (!pinnedMemory)
	{
		free(h_dataIn);
		free(h_dataOut);
	}
	else
	{
		hipHostFree(h_dataIn);
		hipHostFree(h_dataOut);
	}
	hipFree(d_dataIn);
	hipFree(d_dataOut);

	// Print Meassurement Results
	std::cout << "***" << std::endl
			  << "*** Results:" << std::endl
			  << "***    Num Elements: " << numElements << std::endl
			  << "***    Time to Copy to Device: " << 1e3 * memCpyH2DTimer.getTime()
			  	<< " ms" << std::endl
			  << "***    Copy Bandwidth: "
			  	<< 1e-9 * memCpyH2DTimer.getBandwidth(numElements * sizeof(*h_dataIn))
			  	<< " GB/s" << std::endl
			  << "***    Time to Copy from Device: " << 1e3 * memCpyD2HTimer.getTime()
			  	<< " ms" << std::endl
			  << "***    Copy Bandwidth: "
			  	<< 1e-9 * memCpyD2HTimer.getBandwidth(sizeof(*h_dataOut))
				<< " GB/s" << std::endl
			  << "***    Time for Reduction: " << 1e3 * kernelTimer.getTime()
				  << " ms" << std::endl
			  << "***" << std::endl;

	return 0;
}

void
printHelp(char * argv)
{
	std::cout << "Help:" << std::endl
			  << "  Usage: " << std::endl
			  << "  " << argv << " [-p] [-s <num-elements>] [-t <threads_per_block>]"
			  	<< std::endl
			  << "" << std::endl
			  << "  -p|--pinned-memory" << std::endl
			  << "	Use pinned Memory instead of pageable memory" << std::endl
			  << "" << std::endl
			  << "  -s <num-elements>|--size <num-elements>" << std::endl
			  << "	The size of the Matrix" << std::endl
			  << "" << std::endl
			  << "  -t <threads_per_block>|--threads-per-block <threads_per_block>"
			  	<< std::endl
			  << "	The number of threads per block" << std::endl
			  << "" << std::endl;
}
